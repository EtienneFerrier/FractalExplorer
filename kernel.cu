#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <chrono>
#include <SDL2/SDL.h>

#include "Parametres.hpp"
#include "Pixel.hpp"
#include "Mandel.hpp"
#include "Events.hpp"
#include "Affichage.hpp"
#include "BigFloat.hpp"



//__global__ void computeMandel_GPU(uint32_t* result, float xCenter, float yCenter, float scale);

using namespace std;





int affichageGPU(Affichage* disp);


//int main(int argc, char** argv)
//{
//	BigFloat a, b, c, d;
//	a.base = -1;
//	b.base = -1;
//	a[0] = -1;
//	b[0] = -1;
//	a[1] = 0;
//	b[1] = 0;
//	a[2] = 0;
//	b[2] = 0;
//	a[3] = 0;
//	b[3] = 0;
//	BigFloat::mult(a, b, c);
//	BigFloat::mult(a, a, d);
//	a.display();
//	b.display();
//	c.display();
//	cout << "a = " << a.base << ", " << a[0] << " " << a[1] << " " << a[2] << " " << a[3] << endl;
//	cout << "b = " << b.base << ", " << b[0] << " " << b[1] << " " << b[2] << " " << b[3] << endl;
//	cout << "c = " << c.base << ", " << c[0] << " " << c[1] << " " << c[2] << " " << c[3] << endl;
//	cout << "d = " << d.base << ", " << d[0] << " " << d[1] << " " << d[2] << " " << d[3] << endl;
//	while (1);
//	return 0;
//}

int main(int argc, char** argv)
{
	Affichage display;
	if(display.initSDLAffichage() < 0)
		return 0;

	/* Calcul de la fractale */ 
	//Mandelbrot::computeMandel(display.pixels, WIDTH, HEIGHT, display.center, display.scale);

	affichageGPU(&display);
	

	/* Affichage de la fractale */
	//display.dessin();

	/* Boucle des evenements */
	bool quit = false;
	SDL_Event event;

	while (!quit)
	{
		SDL_WaitEvent(&event);
		bool buttonDown;
		switch (event.type)
		{
		case SDL_MOUSEBUTTONDOWN:
			switch (event.button.button)
			{
			case SDL_BUTTON_LEFT:
				buttonDown = true;
				Events::clicGauche(event, &display);
				while (buttonDown)
				{
					SDL_PumpEvents();
					
					if (SDL_GetMouseState(&(event.button.x), &(event.button.y)) & SDL_BUTTON(SDL_BUTTON_LEFT)) {
						Events::clicGauche(event, &display);
					} else {
						buttonDown = false;
					}

				}
				break;
			case SDL_BUTTON_RIGHT:
				buttonDown = true;
				Events::clicDroit(event, &display);
				while (buttonDown)
				{
					SDL_PumpEvents();

					if (SDL_GetMouseState(&(event.button.x), &(event.button.y)) & SDL_BUTTON(SDL_BUTTON_RIGHT)) {
						Events::clicDroit(event, &display);
					}
					else {
						buttonDown = false;
					}

				}
				break;
			default:
				SDL_ShowSimpleMessageBox(0, "Mouse", "Some other button was pressed!", display.win);
				break;
			}
			break;
		case SDL_QUIT:
			quit = true;
			break;
		}

	}

	return 0;
}

