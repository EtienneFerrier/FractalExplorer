#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <chrono>
#include <SDL2/SDL.h>

#include "Parametres.hpp"
#include "Pixel.hpp"
#include "Mandel.hpp"
#include "Events.hpp"
#include "Affichage.hpp"
#include "BigFloat.hpp"



//__global__ void computeMandel_GPU(uint32_t* result, float xCenter, float yCenter, float scale);

using namespace std;





int affichageGPU(Affichage* disp);


int main(int argc, char** argv)
{
	BigFloat big;
	big.decimals[1] = 1;
	big.display();
	while (1);
	return 0;
}

//int main(int argc, char** argv)
//{
//	Affichage display;
//	if(display.initSDLAffichage() < 0)
//		return 0;
//
//	/* Calcul de la fractale */ 
//	//Mandelbrot::computeMandel(display.pixels, WIDTH, HEIGHT, display.center, display.scale);
//
//	affichageGPU(&display);
//	
//
//	/* Affichage de la fractale */
//	//display.dessin();
//
//	/* Boucle des evenements */
//	bool quit = false;
//	SDL_Event event;
//
//	while (!quit)
//	{
//		SDL_WaitEvent(&event);
//		bool buttonDown;
//		switch (event.type)
//		{
//		case SDL_MOUSEBUTTONDOWN:
//			switch (event.button.button)
//			{
//			case SDL_BUTTON_LEFT:
//				buttonDown = true;
//				Events::clicGauche(event, &display);
//				while (buttonDown)
//				{
//					SDL_PumpEvents();
//					
//					if (SDL_GetMouseState(&(event.button.x), &(event.button.y)) & SDL_BUTTON(SDL_BUTTON_LEFT)) {
//						Events::clicGauche(event, &display);
//					} else {
//						buttonDown = false;
//					}
//
//				}
//				break;
//			case SDL_BUTTON_RIGHT:
//				buttonDown = true;
//				Events::clicDroit(event, &display);
//				while (buttonDown)
//				{
//					SDL_PumpEvents();
//
//					if (SDL_GetMouseState(&(event.button.x), &(event.button.y)) & SDL_BUTTON(SDL_BUTTON_RIGHT)) {
//						Events::clicDroit(event, &display);
//					}
//					else {
//						buttonDown = false;
//					}
//
//				}
//				break;
//			default:
//				SDL_ShowSimpleMessageBox(0, "Mouse", "Some other button was pressed!", display.win);
//				break;
//			}
//			break;
//		case SDL_QUIT:
//			quit = true;
//			break;
//		}
//
//	}
//
//	return 0;
//}

