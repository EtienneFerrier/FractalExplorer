#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <sstream>
#include <chrono>
#include <SDL2/SDL.h>

#include "Parametres.hpp"
#include "Pixel.hpp"
#include "Mandel.hpp"
#include "Events.hpp"
#include "Affichage.hpp"

using namespace std;

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}



//int main(int argc, char* argv[])
//{
//	//const int arraySize = 5;
//	//const int a[arraySize] = { 1, 2, 3, 4, 5 };
//	//const int b[arraySize] = { 10, 20, 30, 40, 50 };
//	//int c[arraySize] = { 0 };
//
//	//// Add vectors in parallel.
//	//hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "addWithCuda failed!");
//	//	return 1;
//	//}
//
//	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
//	//	c[0], c[1], c[2], c[3], c[4]);
//
//	//// hipDeviceReset must be called before exiting in order for profiling and
//	//// tracing tools such as Nsight and Visual Profiler to show complete traces.
//	//cudaStatus = hipDeviceReset();
//	//if (cudaStatus != hipSuccess) {
//	//	fprintf(stderr, "hipDeviceReset failed!");
//	//	return 1;
//	//}
//
//	//============================================================ Partie DSL =========================================
//	// Notre fen�tre
//
//	SDL_Window* fenetre(0);
//	SDL_Event evenements;
//	bool terminer(false);
//
//
//	// Initialisation de la SDL
//
//	if (SDL_Init(SDL_INIT_VIDEO) < 0)
//	{
//		std::cout << "Erreur lors de l'initialisation de la SDL : " << SDL_GetError() << std::endl;
//		SDL_Quit();
//
//		return -1;
//	}
//
//
//	// Cr�ation de la fen�tre
//
//	cout << "Go Fenetre" << endl;
//
//	fenetre = SDL_CreateWindow("Test SDL 2.0", SDL_WINDOWPOS_CENTERED, SDL_WINDOWPOS_CENTERED, 800, 600, SDL_WINDOW_SHOWN);
//
//	SDL_Surface* surface = SDL_GetWindowSurface(fenetre);
//
//	const SDL_PixelFormat* format = surface->format;
//	
//	Uint32 pixel = SDL_MapRGB(format, 100, 100, 100);
//
//	//Uint8 r, g, b, a;
//
//
//	//SDL_LockSurface(surface); /*On bloque la surface*/
//
//


int main(int argc, char** argv)
{
	Affichage display;
	if(display.initSDLAffichage() < 0)
		return 0;

	/* Calcul de la fractale */ 
	Mandelbrot::computeMandel(display.pixels, WIDTH, HEIGHT, display.center, display.scale);

	/* Affichage de la fractale */
	display.dessin();

	/* Boucle des evenements */
	bool quit = false;
	SDL_Event event;

	while (!quit)
	{
		SDL_WaitEvent(&event);

		switch (event.type)
		{
		case SDL_MOUSEBUTTONDOWN:
			switch (event.button.button)
			{
			case SDL_BUTTON_LEFT:
				Events::clicGauche(event, &display);
				break;
			case SDL_BUTTON_RIGHT:
				Events::clicDroit(event, &display);
				break;
			default:
				SDL_ShowSimpleMessageBox(0, "Mouse", "Some other button was pressed!", display.win);
				break;
			}
			break;
		case SDL_QUIT:
			quit = true;
			break;
		}

	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	int *dev_a = 0;
	int *dev_b = 0;
	int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	addKernel<<< 1, size>>>(dev_c, dev_a, dev_b);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}